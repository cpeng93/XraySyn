#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <vector>

using namespace std;
using scalar_t = float;

const int MAX_BATCH_SIZE = 64;
__constant__ scalar_t RAY_MAT[9 * MAX_BATCH_SIZE];
__constant__ scalar_t RAY[3 * MAX_BATCH_SIZE];


__global__ void dp_trilinear_cuda_forward_kernel(
  scalar_t* __restrict__ volume,
  scalar_t* __restrict__ projection,
  int nx, int ny, int nz,
  int dh, int dw, int bs,
  scalar_t step, scalar_t vx, scalar_t vy, scalar_t vz
) {
  int udx = threadIdx.x + blockIdx.x * blockDim.x;
  int vdx = threadIdx.y + blockIdx.y * blockDim.y;
  int idx = blockIdx.z * dw * dh + udx * dw + vdx;

  if (udx >= dh || vdx >= dw) { return; }

  scalar_t* ray_mat = RAY_MAT + (blockIdx.z % bs) * 9;
  scalar_t* ray = RAY + (blockIdx.z % bs) * 3;
  scalar_t u = (scalar_t) udx + 0.5f;
  scalar_t v = (scalar_t) vdx + 0.5f;
  scalar_t sx = ray[0];
  scalar_t sy = ray[1];
  scalar_t sz = ray[2];

  // compute ray direction
  scalar_t rx = ray_mat[2] + v * ray_mat[1] + u * ray_mat[0];
  scalar_t ry = ray_mat[5] + v * ray_mat[4] + u * ray_mat[3];
  scalar_t rz = ray_mat[8] + v * ray_mat[7] + u * ray_mat[6];

  // normalize ray direction
  scalar_t nf = 1.0f / (sqrt((rx * rx) + (ry * ry) + (rz * rz)));
  rx *= nf;
  ry *= nf;
  rz *= nf;

  //calculate projections
  // Step 1: compute alpha value at entry and exit point of the volume
  scalar_t minAlpha, maxAlpha;
  scalar_t alpha0, alpha1;

  minAlpha = 0.0f;
  maxAlpha = INFINITY;

  if (0.0f != rx)
  {
    alpha0 = -sx / rx;
    alpha1 = (nx - sx) / rx;
    minAlpha = fmin(alpha0, alpha1);
    maxAlpha = fmax(alpha0, alpha1);
  } else if (0.0f > sx || sx > nx) {
    return;
  }

  if (0.0f != ry)
  {
    alpha0 = -sy / ry;
    alpha1 = (ny - sy) / ry;
    minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
    maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
  } else if (0.0f > sy || sy > ny) {
    return;
  }

  if (0.0f != rz)
  {
    alpha0 = - sz / rz;
    alpha1 = (nz - sz) / rz;
    minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
    maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
  } else if (0.0f > sz || sz > nz) {
    return;
  }

  // Step 2: Cast ray if it intersects the volume
  // Trapezoidal rule (interpolating function = piecewise linear func)
  scalar_t temp;
  scalar_t px, py, pz;
  scalar_t v000, v100, v010, v001, v110, v101, v011, v111;
  scalar_t tx0, ty0, tz0, tx1, ty1, tz1;
  int ix0, iy0, iz0, ix1, iy1, iz1;
  int nyz = ny * nz;

  // sx = sx - 0.5f;
  // sy = sy - 0.5f;
  // sz = sz - 0.5f;
  volume = volume + nx * nyz * blockIdx.z;

  // Entrance boundary
  // In CUDA, voxel centers are located at (xx.5, xx.5, xx.5),
  // whereas, SwVolume has voxel centers at integers.
  // For the initial interpolated value, only a half stepsize is
  //  considered in the computation.
  // bool check = minAlpha < maxAlpha;

  // Mid segments
  while (minAlpha < maxAlpha)
  {
    px = sx + minAlpha * rx;
    py = sy + minAlpha * ry;
    pz = sz + minAlpha * rz;

    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;

      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;

      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      v000 = volume[nyz * ix0 + nz * iy0 + iz0];
      v100 = volume[nyz * ix1 + nz * iy0 + iz0];
      v010 = volume[nyz * ix0 + nz * iy1 + iz0];
      v001 = volume[nyz * ix0 + nz * iy0 + iz1];
      v110 = volume[nyz * ix1 + nz * iy1 + iz0];
      v101 = volume[nyz * ix1 + nz * iy0 + iz1];
      v011 = volume[nyz * ix0 + nz * iy1 + iz1];
      v111 = volume[nyz * ix1 + nz * iy1 + iz1];

      temp = tx1 * ty1 * tz1 * v000 +
             tx0 * ty1 * tz1 * v100 +
             tx1 * ty0 * tz1 * v010 +
             tx1 * ty1 * tz0 * v001 +
             tx0 * ty0 * tz1 * v110 +
             tx0 * ty1 * tz0 * v101 +
             tx1 * ty0 * tz0 * v011 +
             tx0 * ty0 * tz0 * v111;
      projection[idx] += temp;
    }
    minAlpha += step;
  }

  // Scaling by stepsize;
  projection[idx] *= step;

  if (check) {
    minAlpha -= step;
    scalar_t lastStepsize = maxAlpha - minAlpha;
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      projection[idx] -= 0.5f * step * temp;
      projection[idx] += 0.5f * lastStepsize * temp;
    }
    px = sx + maxAlpha * rx;
    py = sy + maxAlpha * ry;
    pz = sz + maxAlpha * rz;

    // The last segment of the line integral takes care of the
    // varying length.
    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;

      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;

      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      v000 = volume[nyz * ix0 + nz * iy0 + iz0];
      v100 = volume[nyz * ix1 + nz * iy0 + iz0];
      v010 = volume[nyz * ix0 + nz * iy1 + iz0];
      v001 = volume[nyz * ix0 + nz * iy0 + iz1];
      v110 = volume[nyz * ix1 + nz * iy1 + iz0];
      v101 = volume[nyz * ix1 + nz * iy0 + iz1];
      v011 = volume[nyz * ix0 + nz * iy1 + iz1];
      v111 = volume[nyz * ix1 + nz * iy1 + iz1];

      temp = tx1 * ty1 * tz1 * v000 +
             tx0 * ty1 * tz1 * v100 +
             tx1 * ty0 * tz1 * v010 +
             tx1 * ty1 * tz0 * v001 +
             tx0 * ty0 * tz1 * v110 +
             tx0 * ty1 * tz0 * v101 +
             tx1 * ty0 * tz0 * v011 +
             tx0 * ty0 * tz0 * v111;
      projection[idx] += 0.5f * lastStepsize * temp;
    }
  }
  projection[idx] *= sqrt((rx * vx)*(rx * vx) + (ry * vy)*(ry * vy) + (rz * vz)*(rz * vz));
}

__global__ void dp_trilinear_cuda_backward_kernel(
  scalar_t* __restrict__ volume,
  scalar_t* __restrict__ projection,
  int nx, int ny, int nz,
  int dh, int dw, int bs,
  scalar_t step, scalar_t vx, scalar_t vy, scalar_t vz
) {
  int udx = threadIdx.x + blockIdx.x * blockDim.x;
  int vdx = threadIdx.y + blockIdx.y * blockDim.y;
  int idx = blockIdx.z * dw * dh + udx * dw + vdx;

  if (udx >= dh || vdx >= dw) { return; }

  scalar_t* ray_mat = RAY_MAT + (blockIdx.z % bs) * 9;
  scalar_t* ray = RAY + (blockIdx.z % bs) * 3;
  scalar_t u = (scalar_t) udx + 0.5f;
  scalar_t v = (scalar_t) vdx + 0.5f;
  scalar_t sx = ray[0];
  scalar_t sy = ray[1];
  scalar_t sz = ray[2];

  // compute ray direction
  scalar_t rx = ray_mat[2] + v * ray_mat[1] + u * ray_mat[0];
  scalar_t ry = ray_mat[5] + v * ray_mat[4] + u * ray_mat[3];
  scalar_t rz = ray_mat[8] + v * ray_mat[7] + u * ray_mat[6];

  // normalize ray direction
  scalar_t nf = 1.0f / (sqrt((rx * rx) + (ry * ry) + (rz * rz)));
  rx *= nf;
  ry *= nf;
  rz *= nf;

  //calculate projections
  // Step 1: compute alpha value at entry and exit point of the volume
  scalar_t minAlpha, maxAlpha;
  scalar_t alpha0, alpha1;

  minAlpha = 0.0f;
  maxAlpha = INFINITY;

  if (0.0f != rx)
  {
    alpha0 = -sx / rx;
    alpha1 = (nx - sx) / rx;
    minAlpha = fmin(alpha0, alpha1);
    maxAlpha = fmax(alpha0, alpha1);
  } else if (0.0f > sx || sx > nx) {
    return;
  }

  if (0.0f != ry)
  {
    alpha0 = -sy / ry;
    alpha1 = (ny - sy) / ry;
    minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
    maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
  } else if (0.0f > sy || sy > ny) {
    return;
  }

  if (0.0f != rz)
  {
    alpha0 = - sz / rz;
    alpha1 = (nz - sz) / rz;
    minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
    maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
  } else if (0.0f > sz || sz > nz) {
    return;
  }

  // Step 2: Cast ray if it intersects the volume

  // Trapezoidal rule (interpolating function = piecewise linear func)
  scalar_t px, py, pz;
  scalar_t tx0, ty0, tz0, tx1, ty1, tz1;
  scalar_t temp;
  scalar_t proj;
  int ix0, iy0, iz0, ix1, iy1, iz1;
  int iv000, iv100, iv010, iv001, iv110, iv101, iv011, iv111;
  int nyz = ny * nz;

  sx = sx - 0.5f;
  sy = sy - 0.5f;
  sz = sz - 0.5f;
  volume = volume + nx * nyz * blockIdx.z;
  proj = projection[idx] * step * sqrt((rx * vx)*(rx * vx) + (ry * vy)*(ry * vy) + (rz * vz)*(rz * vz));

  // Entrance boundary
  // In CUDA, voxel centers are located at (xx.5, xx.5, xx.5),
  // whereas, SwVolume has voxel centers at integers.
  // For the initial interpolated value, only a half stepsize is
  //  considered in the computation.
  bool check = minAlpha < maxAlpha;
  if (minAlpha < maxAlpha) {
    px = sx + minAlpha * rx;
    py = sy + minAlpha * ry;
    pz = sz + minAlpha * rz;

    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;
  
      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;
  
      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      iv000 = nyz * ix0 + nz * iy0 + iz0;
      iv100 = nyz * ix1 + nz * iy0 + iz0;
      iv010 = nyz * ix0 + nz * iy1 + iz0;
      iv001 = nyz * ix0 + nz * iy0 + iz1;
      iv110 = nyz * ix1 + nz * iy1 + iz0;
      iv101 = nyz * ix1 + nz * iy0 + iz1;
      iv011 = nyz * ix0 + nz * iy1 + iz1;
      iv111 = nyz * ix1 + nz * iy1 + iz1;

      temp = 0.5 * proj;
      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * temp);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * temp);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * temp);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * temp);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * temp);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * temp);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * temp);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * temp);
    }
    minAlpha += step;
  }

  // Mid segments
  while (minAlpha < maxAlpha)
  {
    px = sx + minAlpha * rx;
    py = sy + minAlpha * ry;
    pz = sz + minAlpha * rz;

    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;
  
      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;
  
      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      iv000 = nyz * ix0 + nz * iy0 + iz0;
      iv100 = nyz * ix1 + nz * iy0 + iz0;
      iv010 = nyz * ix0 + nz * iy1 + iz0;
      iv001 = nyz * ix0 + nz * iy0 + iz1;
      iv110 = nyz * ix1 + nz * iy1 + iz0;
      iv101 = nyz * ix1 + nz * iy0 + iz1;
      iv011 = nyz * ix0 + nz * iy1 + iz1;
      iv111 = nyz * ix1 + nz * iy1 + iz1;

      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * proj);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * proj);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * proj);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * proj);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * proj);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * proj);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * proj);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * proj);
    }
    minAlpha += step;
  }

  if (check) {
    minAlpha -= step;
    scalar_t c = (maxAlpha - minAlpha) / step;
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      temp = 0.5f * (c - 1.0f) * proj;
      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * temp);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * temp);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * temp);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * temp);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * temp);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * temp);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * temp);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * temp);
    }
    px = sx + maxAlpha * rx;
    py = sy + maxAlpha * ry;
    pz = sz + maxAlpha * rz;

    // The last segment of the line integral takes care of the
    // varying length.
    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;
  
      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;
  
      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      iv000 = nyz * ix0 + nz * iy0 + iz0;
      iv100 = nyz * ix1 + nz * iy0 + iz0;
      iv010 = nyz * ix0 + nz * iy1 + iz0;
      iv001 = nyz * ix0 + nz * iy0 + iz1;
      iv110 = nyz * ix1 + nz * iy1 + iz0;
      iv101 = nyz * ix1 + nz * iy0 + iz1;
      iv011 = nyz * ix0 + nz * iy1 + iz1;
      iv111 = nyz * ix1 + nz * iy1 + iz1;

      temp = 0.5f * c * proj;
      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * temp);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * temp);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * temp);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * temp);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * temp);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * temp);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * temp);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * temp);
    }
  }
}

__global__ void dp_backproject_trilinear_cuda_forward(
  scalar_t* __restrict__ volume,
  scalar_t* __restrict__ projection,
  int nx, int ny, int nz,
  int dh, int dw, int bs,
  scalar_t step, scalar_t vx, scalar_t vy, scalar_t vz
) {
  int udx = threadIdx.x + blockIdx.x * blockDim.x;
  int vdx = threadIdx.y + blockIdx.y * blockDim.y;
  int idx = blockIdx.z * dw * dh + udx * dw + vdx;

  if (udx >= dh || vdx >= dw) { return; }

  scalar_t* ray_mat = RAY_MAT + (blockIdx.z % bs) * 9;
  scalar_t* ray = RAY + (blockIdx.z % bs) * 3;
  scalar_t u = (scalar_t) udx + 0.5f;
  scalar_t v = (scalar_t) vdx + 0.5f;
  scalar_t sx = ray[0];
  scalar_t sy = ray[1];
  scalar_t sz = ray[2];

  // compute ray direction
  scalar_t rx = ray_mat[2] + v * ray_mat[1] + u * ray_mat[0];
  scalar_t ry = ray_mat[5] + v * ray_mat[4] + u * ray_mat[3];
  scalar_t rz = ray_mat[8] + v * ray_mat[7] + u * ray_mat[6];

  // normalize ray direction
  scalar_t nf = 1.0f / (sqrt((rx * rx) + (ry * ry) + (rz * rz)));
  rx *= nf;
  ry *= nf;
  rz *= nf;

  //calculate projections
  // Step 1: compute alpha value at entry and exit point of the volume
  scalar_t minAlpha, maxAlpha;
  scalar_t alpha0, alpha1;

  minAlpha = 0.0f;
  maxAlpha = INFINITY;

  if (0.0f != rx)
  {
    alpha0 = -sx / rx;
    alpha1 = (nx - sx) / rx;
    minAlpha = fmin(alpha0, alpha1);
    maxAlpha = fmax(alpha0, alpha1);
  } else if (0.0f > sx || sx > nx) {
    return;
  }

  if (0.0f != ry)
  {
    alpha0 = -sy / ry;
    alpha1 = (ny - sy) / ry;
    minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
    maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
  } else if (0.0f > sy || sy > ny) {
    return;
  }

  if (0.0f != rz)
  {
    alpha0 = - sz / rz;
    alpha1 = (nz - sz) / rz;
    minAlpha = fmax(minAlpha, fmin(alpha0, alpha1));
    maxAlpha = fmin(maxAlpha, fmax(alpha0, alpha1));
  } else if (0.0f > sz || sz > nz) {
    return;
  }

  // Step 2: Cast ray if it intersects the volume

  // Trapezoidal rule (interpolating function = piecewise linear func)
  scalar_t px, py, pz;
  scalar_t tx0, ty0, tz0, tx1, ty1, tz1;
  scalar_t temp;
  scalar_t proj;
  int ix0, iy0, iz0, ix1, iy1, iz1;
  int iv000, iv100, iv010, iv001, iv110, iv101, iv011, iv111;
  int nyz = ny * nz;

  sx = sx - 0.5f;
  sy = sy - 0.5f;
  sz = sz - 0.5f;
  volume = volume + nx * nyz * blockIdx.z;


  scalar_t total_steps = (maxAlpha - minAlpha)/step;


  proj = projection[idx] / sqrt((rx * vx)*(rx * vx) + (ry * vy)*(ry * vy) + (rz * vz)*(rz * vz));

  proj = proj / total_steps;

  // Entrance boundary
  // In CUDA, voxel centers are located at (xx.5, xx.5, xx.5),
  // whereas, SwVolume has voxel centers at integers.
  // For the initial interpolated value, only a half stepsize is
  //  considered in the computation.
  bool check = minAlpha < maxAlpha;
  if (minAlpha < maxAlpha) {
    px = sx + minAlpha * rx;
    py = sy + minAlpha * ry;
    pz = sz + minAlpha * rz;

    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;

      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;

      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      iv000 = nyz * ix0 + nz * iy0 + iz0;
      iv100 = nyz * ix1 + nz * iy0 + iz0;
      iv010 = nyz * ix0 + nz * iy1 + iz0;
      iv001 = nyz * ix0 + nz * iy0 + iz1;
      iv110 = nyz * ix1 + nz * iy1 + iz0;
      iv101 = nyz * ix1 + nz * iy0 + iz1;
      iv011 = nyz * ix0 + nz * iy1 + iz1;
      iv111 = nyz * ix1 + nz * iy1 + iz1;

      temp = 0.5 * proj;
      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * temp);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * temp);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * temp);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * temp);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * temp);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * temp);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * temp);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * temp);
    }
    minAlpha += step;
  }

  // Mid segments
  while (minAlpha < maxAlpha)
  {
    px = sx + minAlpha * rx;
    py = sy + minAlpha * ry;
    pz = sz + minAlpha * rz;

    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;

      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;

      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      iv000 = nyz * ix0 + nz * iy0 + iz0;
      iv100 = nyz * ix1 + nz * iy0 + iz0;
      iv010 = nyz * ix0 + nz * iy1 + iz0;
      iv001 = nyz * ix0 + nz * iy0 + iz1;
      iv110 = nyz * ix1 + nz * iy1 + iz0;
      iv101 = nyz * ix1 + nz * iy0 + iz1;
      iv011 = nyz * ix0 + nz * iy1 + iz1;
      iv111 = nyz * ix1 + nz * iy1 + iz1;

      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * proj);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * proj);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * proj);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * proj);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * proj);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * proj);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * proj);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * proj);
    }
    minAlpha += step;
  }

  if (check) {
    minAlpha -= step;
    scalar_t c = (maxAlpha - minAlpha) / step;
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      temp = 0.5f * (c - 1.0f) * proj;
      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * temp);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * temp);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * temp);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * temp);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * temp);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * temp);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * temp);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * temp);
    }
    px = sx + maxAlpha * rx;
    py = sy + maxAlpha * ry;
    pz = sz + maxAlpha * rz;

    // The last segment of the line integral takes care of the
    // varying length.
    ix0 = int(px);
    iy0 = int(py);
    iz0 = int(pz);
    if (ix0 >= 0 && ix0 < nx - 1 && iy0 >= 0 && iy0 < ny - 1 && iz0 >= 0 && iz0 < nz - 1) {
      tx0 = px - ix0;
      ty0 = py - iy0;
      tz0 = pz - iz0;

      ix1 = ix0 + 1;
      iy1 = iy0 + 1;
      iz1 = iz0 + 1;

      tx1 = ix1 - px;
      ty1 = iy1 - py;
      tz1 = iz1 - pz;

      iv000 = nyz * ix0 + nz * iy0 + iz0;
      iv100 = nyz * ix1 + nz * iy0 + iz0;
      iv010 = nyz * ix0 + nz * iy1 + iz0;
      iv001 = nyz * ix0 + nz * iy0 + iz1;
      iv110 = nyz * ix1 + nz * iy1 + iz0;
      iv101 = nyz * ix1 + nz * iy0 + iz1;
      iv011 = nyz * ix0 + nz * iy1 + iz1;
      iv111 = nyz * ix1 + nz * iy1 + iz1;

      temp = 0.5f * c * proj;
      atomicAdd(volume + iv000, tx1 * ty1 * tz1 * temp);
      atomicAdd(volume + iv100, tx0 * ty1 * tz1 * temp);
      atomicAdd(volume + iv010, tx1 * ty0 * tz1 * temp);
      atomicAdd(volume + iv001, tx1 * ty1 * tz0 * temp);
      atomicAdd(volume + iv110, tx0 * ty0 * tz1 * temp);
      atomicAdd(volume + iv101, tx0 * ty1 * tz0 * temp);
      atomicAdd(volume + iv011, tx1 * ty0 * tz0 * temp);
      atomicAdd(volume + iv111, tx0 * ty0 * tz0 * temp);
    }
  }
}

torch::Tensor dp_trilinear_cuda_forward(
  torch::Tensor volume,
  torch::Tensor detector_shape,
  torch::Tensor ray_mat,
  torch::Tensor ray,
  torch::Tensor step_size,
  torch::Tensor voxel_size) {

  torch::Tensor projection; // projection tensor
  scalar_t *volume_ptr; // volume ptr
  scalar_t *proj_ptr; // projection ptr
  scalar_t *ray_mat_ptr; // ray direction matrix ptr
  scalar_t *ray_ptr; // ray direction ptr
  scalar_t step; // step size
  scalar_t vx, vy, vz; // voxel size
  int batch_size; // batch size
  int dh, dw; // detector shape
  int nx, ny, nz; // volume shape
  int nc; // number of channels

  // accessors
  auto d_a = detector_shape.accessor<int,1>();
  auto v_a = voxel_size.accessor<scalar_t,1>();
  auto s_a = step_size.accessor<scalar_t,1>();

  batch_size = volume.size(0); AT_ASSERT(batch_size <= MAX_BATCH_SIZE);
  nc = volume.size(1);
  nx = volume.size(2);
  ny = volume.size(3);
  nz = volume.size(4);

  dh = d_a[0];
  dw = d_a[1];
  vx = v_a[0];
  vy = v_a[1];
  vz = v_a[2];
  step = s_a[0];

  projection = torch::zeros({batch_size, nc, dh, dw}, torch::TensorOptions().
    dtype(volume.dtype()).device(volume.device()));
  volume_ptr = volume.data<scalar_t>();
  proj_ptr = projection.data<scalar_t>();
  ray_mat_ptr = ray_mat.data<scalar_t>();
  ray_ptr = ray.data<scalar_t>();

  // move data to constant memory (which supports the fastest memory access)
  hipMemcpyToSymbol(HIP_SYMBOL(RAY_MAT), ray_mat_ptr, sizeof(scalar_t) * batch_size * 9);
  hipMemcpyToSymbol(HIP_SYMBOL(RAY), ray_ptr, sizeof(scalar_t) * batch_size * 3);

  const int nblock_h = (dh + 16 - 1) / 16;
  const int nblock_w = (dw + 16 - 1) / 16;
  const dim3 block_size(16, 16, 1);
  const dim3 grid_size(nblock_h, nblock_w, batch_size * nc);

  dp_trilinear_cuda_forward_kernel<<<grid_size, block_size>>>(
    volume_ptr, proj_ptr, nx, ny, nz, dh, dw, batch_size, step, vx, vy, vz
  );
  return projection;
}

torch::Tensor dp_trilinear_cuda_backward(
  torch::Tensor projection,
  torch::Tensor volume_shape,
  torch::Tensor ray_mat,
  torch::Tensor ray,
  torch::Tensor step_size,
  torch::Tensor voxel_size) {

  torch::Tensor volume; // volume tensor
  scalar_t *volume_ptr; // volume ptr
  scalar_t *proj_ptr; // projection ptr
  scalar_t *ray_mat_ptr; // ray direction matrix ptr
  scalar_t *ray_ptr; // ray direction ptr
  scalar_t step; // step size
  scalar_t vx, vy, vz; // voxel size
  int batch_size; // batch size
  int dh, dw; // detector shape
  int nx, ny, nz; // volume shape
  int nc; // number of channels

  // accessors
  auto n_a = volume_shape.accessor<int,1>();
  auto v_a = voxel_size.accessor<scalar_t,1>();
  auto s_a = step_size.accessor<scalar_t,1>();

  batch_size = projection.size(0); AT_ASSERT(batch_size <= MAX_BATCH_SIZE);
  nc = projection.size(1);
  dh = projection.size(2);
  dw = projection.size(3);

  nx = n_a[0];
  ny = n_a[1];
  nz = n_a[2];
  vx = v_a[0];
  vy = v_a[1];
  vz = v_a[2];
  step = s_a[0];

  volume = torch::zeros({batch_size, nc, nx, ny, nz}, torch::TensorOptions().
    dtype(projection.dtype()).device(projection.device()));
  volume_ptr = volume.data<scalar_t>();
  proj_ptr = projection.data<scalar_t>();
  ray_mat_ptr = ray_mat.data<scalar_t>();
  ray_ptr = ray.data<scalar_t>();

  // move data to constant memory (which supports the fastest memory access)
  hipMemcpyToSymbol(HIP_SYMBOL(RAY_MAT), ray_mat_ptr, sizeof(scalar_t) * batch_size * 9);
  hipMemcpyToSymbol(HIP_SYMBOL(RAY), ray_ptr, sizeof(scalar_t) * batch_size * 3);

  const int nblock_h = (dh + 16 - 1) / 16;
  const int nblock_w = (dw + 16 - 1) / 16;
  const dim3 block_size(16, 16, 1);
  const dim3 grid_size(nblock_h, nblock_w, batch_size * nc);

  dp_trilinear_cuda_backward_kernel<<<grid_size, block_size>>>(
    volume_ptr, proj_ptr, nx, ny, nz, dh, dw, batch_size, step, vx, vy, vz
  );
  return volume;
}

torch::Tensor dp_backproject_trilinear_cuda_forward(
  torch::Tensor projection,
  torch::Tensor volume_shape,
  torch::Tensor ray_mat,
  torch::Tensor ray,
  torch::Tensor step_size,
  torch::Tensor voxel_size) {

  torch::Tensor volume; // volume tensor
  scalar_t *volume_ptr; // volume ptr
  scalar_t *proj_ptr; // projection ptr
  scalar_t *ray_mat_ptr; // ray direction matrix ptr
  scalar_t *ray_ptr; // ray direction ptr
  scalar_t step; // step size
  scalar_t vx, vy, vz; // voxel size
  int batch_size; // batch size
  int dh, dw; // detector shape
  int nx, ny, nz; // volume shape
  int nc; // number of channels

  // accessors
  auto n_a = volume_shape.accessor<int,1>();
  auto v_a = voxel_size.accessor<scalar_t,1>();
  auto s_a = step_size.accessor<scalar_t,1>();

  batch_size = projection.size(0); AT_ASSERT(batch_size <= MAX_BATCH_SIZE);
  nc = projection.size(1);
  dh = projection.size(2);
  dw = projection.size(3);

  nx = n_a[0];
  ny = n_a[1];
  nz = n_a[2];
  vx = v_a[0];
  vy = v_a[1];
  vz = v_a[2];
  step = s_a[0];

  volume = torch::zeros({batch_size, nc, nx, ny, nz}, torch::TensorOptions().
    dtype(projection.dtype()).device(projection.device()));
  volume_ptr = volume.data<scalar_t>();
  proj_ptr = projection.data<scalar_t>();
  ray_mat_ptr = ray_mat.data<scalar_t>();
  ray_ptr = ray.data<scalar_t>();

  // move data to constant memory (which supports the fastest memory access)
  hipMemcpyToSymbol(HIP_SYMBOL(RAY_MAT), ray_mat_ptr, sizeof(scalar_t) * batch_size * 9);
  hipMemcpyToSymbol(HIP_SYMBOL(RAY), ray_ptr, sizeof(scalar_t) * batch_size * 3);

  const int nblock_h = (dh + 16 - 1) / 16;
  const int nblock_w = (dw + 16 - 1) / 16;
  const dim3 block_size(16, 16, 1);
  const dim3 grid_size(nblock_h, nblock_w, batch_size * nc);

  dp_backproject_trilinear_cuda_forward<<<grid_size, block_size>>>(
    volume_ptr, proj_ptr, nx, ny, nz, dh, dw, batch_size, step, vx, vy, vz
  );
  return volume;
}